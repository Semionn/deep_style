#include "hip/hip_runtime.h"
#include <cfloat>
#include "cudarray/common.hpp"
#include "cudarray/nnet/pool_b01.hpp"

namespace cudarray {

// The implementations below are inspired by those found in the Caffe framework

template <typename T>
__global__ void max_pool_b01(int n_threads, const T* imgs,
    int img_h, int img_w, int poolout_h, int poolout_w, int win_h, int win_w,
    int pad_y, int pad_x, int stride_y, int stride_x, T* poolout, int* mask) {
  CUDA_GRID_STRIDE_LOOP(idx, n_threads) {
    int poolout_x = idx % poolout_w;
    int poolout_y = (idx / poolout_w) % poolout_h;
    int n = idx / poolout_w / poolout_h;
    int img_y_start = poolout_y * stride_y - pad_y;
    int img_x_start = poolout_x * stride_x - pad_x;
    int img_y_end = min(img_y_start + win_h, img_h);
    int img_x_end = min(img_x_start + win_w, img_w);
    img_y_start = max(img_y_start, 0);
    img_x_start = max(img_x_start, 0);
    T maxval = -FLT_MAX;
    int maxidx = -1;
    imgs += n * img_h * img_w;
    for (int h = img_y_start; h < img_y_end; ++h) {
      for (int w = img_x_start; w < img_x_end; ++w) {
        if (imgs[h * img_w + w] > maxval) {
          maxidx = h * img_w + w;
          maxval = imgs[maxidx];
        }
      }
    }
    poolout[idx] = maxval;
    mask[idx] = maxidx;
  }
}

template <typename T>
void max_pool_b01(const T* imgs, int n_imgs, int img_h, int img_w, int win_h,
    int win_w, int pad_y, int pad_x, int stride_y, int stride_x, T* poolout,
    int* mask) {
  int poolout_h = (img_h + 2*pad_y - win_h) / stride_y + 1;
  int poolout_w = (img_w + 2*pad_x - win_w) / stride_x + 1;
  int n_threads = n_imgs * poolout_h * poolout_w;
  max_pool_b01<<<cuda_blocks(n_threads), kNumBlockThreads>>>(
    n_threads, imgs, img_h, img_w, poolout_h, poolout_w, win_h, win_w, pad_y,
    pad_x, stride_y, stride_x, poolout, mask);
  CUDA_KERNEL_CHECK;
}

template void max_pool_b01<float>(const float* imgs, int n_imgs, int img_h,
    int img_w, int win_h, int win_w, int pad_y, int pad_x, int stride_y,
    int stride_x, float* poolout, int* mask);



template <typename T>
__global__ void max_pool_b01_bprob(int n_threads, const T* poolout_d,
    const int* mask, int img_h, int img_w, int poolout_h, int poolout_w,
    int win_h, int win_w, int pad_y, int pad_x, int stride_y, int stride_x,
    T* imgs_d) {
  CUDA_GRID_STRIDE_LOOP(idx, n_threads) {
    int img_x = idx % img_w;
    int img_y = (idx / img_w) % img_h;
    int n = idx / img_w / img_h;
    int poolout_y_start = (img_y + pad_y < win_h)
                      ? 0 : (img_y + pad_y - win_h) / stride_y + 1;
    int poolout_y_end = min((img_y + pad_y) / stride_y + 1, poolout_h);
    int poolout_x_start = (img_x + pad_x < win_w)
                      ? 0 : (img_x + pad_x - win_w) / stride_x + 1;
    int poolout_x_end = min((img_x + pad_x) / stride_x + 1, poolout_w);
    int offset = n * poolout_h * poolout_w;
    poolout_d += offset;
    mask += offset;
    T gradient = 0;
    for (int ph = poolout_y_start; ph < poolout_y_end; ++ph) {
      for (int pw = poolout_x_start; pw < poolout_x_end; ++pw) {
        if (mask[ph * poolout_w + pw] == img_y * img_w + img_x) {
          gradient += poolout_d[ph * poolout_w + pw];
        }
      }
    }
    imgs_d[idx] = gradient;
  }
}

template <typename T>
void max_pool_b01_bprob(const T* poolout_d, const int* mask, int n_imgs,
    int img_h, int img_w, int win_h, int win_w, int pad_y, int pad_x,
    int stride_y, int stride_x, T* imgs_d) {
  int poolout_h = (img_h + 2*pad_y - win_h) / stride_y + 1;
  int poolout_w = (img_w + 2*pad_x - win_w) / stride_x + 1;
  int n_threads = n_imgs * img_h * img_w;
  max_pool_b01_bprob<<<cuda_blocks(n_threads), kNumBlockThreads>>>(
    n_threads, poolout_d, mask, img_h, img_w, poolout_h, poolout_w, win_h,
    win_w, pad_y, pad_x, stride_y, stride_x, imgs_d);
  CUDA_KERNEL_CHECK;
}

template void max_pool_b01_bprob(const float* poolout_d, const int* mask,
    int n_imgs, int img_h, int img_w, int win_h, int win_w, int pad_y,
    int pad_x, int stride_y, int stride_x, float* imgs_d);




template <typename T>
__global__ void avg_pool_b01(int n_threads, const T* imgs,
    int img_h, int img_w, int poolout_h, int poolout_w, int win_h, int win_w,
    int pad_y, int pad_x, int stride_y, int stride_x, T* poolout) {
  CUDA_GRID_STRIDE_LOOP(idx, n_threads) {
    int poolout_x = idx % poolout_w;
    int poolout_y = (idx / poolout_w) % poolout_h;
    int n = idx / poolout_w / poolout_h;
    int img_y_start = poolout_y * stride_y - pad_y;
    int img_x_start = poolout_x * stride_x - pad_x;
    int img_y_end = min(img_y_start + win_h, img_h);
    int img_x_end = min(img_x_start + win_w, img_w);
    img_y_start = max(img_y_start, 0);
    img_x_start = max(img_x_start, 0);
    T sum = 0;
    imgs += n * img_h * img_w;
    for (int h = img_y_start; h < img_y_end; ++h) {
      for (int w = img_x_start; w < img_x_end; ++w) {
        sum += imgs[h * img_w + w];
      }
    }
    poolout[idx] = sum / (win_h*win_w);
  }
}

template <typename T>
void avg_pool_b01(const T* imgs, int n_imgs, int img_h, int img_w, int win_h,
    int win_w, int pad_y, int pad_x, int stride_y, int stride_x, T* poolout) {
  int poolout_h = (img_h + 2*pad_y - win_h) / stride_y + 1;
  int poolout_w = (img_w + 2*pad_x - win_w) / stride_x + 1;
  int n_threads = n_imgs * poolout_h * poolout_w;
  avg_pool_b01<<<cuda_blocks(n_threads), kNumBlockThreads>>>(
    n_threads, imgs, img_h, img_w, poolout_h, poolout_w, win_h, win_w, pad_y,
    pad_x, stride_y, stride_x, poolout);
  CUDA_KERNEL_CHECK;
}

template void avg_pool_b01<float>(const float* imgs, int n_imgs, int img_h,
    int img_w, int win_h, int win_w, int pad_y, int pad_x, int stride_y,
    int stride_x, float* poolout);



template <typename T>
__global__ void avg_pool_b01_bprob(int n_threads, const T* poolout_d,
    int img_h, int img_w, int poolout_h, int poolout_w, int win_h, int win_w,
    int pad_y, int pad_x, int stride_y, int stride_x, T* imgs_d) {
  CUDA_GRID_STRIDE_LOOP(idx, n_threads) {
    int img_x = idx % img_w;
    int img_y = (idx / img_w) % img_h;
    int n = idx / img_w / img_h;
    int poolout_y_start = (img_y + pad_y < win_h)
                      ? 0 : (img_y + pad_y - win_h) / stride_y + 1;
    int poolout_y_end = min((img_y + pad_y) / stride_y + 1, poolout_h);
    int poolout_x_start = (img_x + pad_x < win_w)
                      ? 0 : (img_x + pad_x - win_w) / stride_x + 1;
    int poolout_x_end = min((img_x + pad_x) / stride_x + 1, poolout_w);
    int offset = n * poolout_h * poolout_w;
    poolout_d += offset;
    T gradient = 0;
    for (int ph = poolout_y_start; ph < poolout_y_end; ++ph) {
      for (int pw = poolout_x_start; pw < poolout_x_end; ++pw) {
        gradient += poolout_d[ph * poolout_w + pw];
      }
    }
    imgs_d[idx] = gradient / (win_h * win_w);
  }
}

template <typename T>
void avg_pool_b01_bprob(const T* poolout_d, int n_imgs, int img_h, int img_w,
    int win_h, int win_w, int pad_y, int pad_x, int stride_y, int stride_x,
    T* imgs_d) {
  int poolout_h = (img_h + 2*pad_y - win_h) / stride_y + 1;
  int poolout_w = (img_w + 2*pad_x - win_w) / stride_x + 1;
  int n_threads = n_imgs * img_h * img_w;
  avg_pool_b01_bprob<<<cuda_blocks(n_threads), kNumBlockThreads>>>(
    n_threads, poolout_d, img_h, img_w, poolout_h, poolout_w, win_h,
    win_w, pad_y, pad_x, stride_y, stride_x, imgs_d);
  CUDA_KERNEL_CHECK;
}

template void avg_pool_b01_bprob(const float* poolout_d, int n_imgs, int img_h,
    int img_w, int win_h, int win_w, int pad_y, int pad_x, int stride_y,
    int stride_x, float* imgs_d);


}
